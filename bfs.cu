#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <climits>
#include <fstream>
#include <sstream>
#include <chrono>

using namespace std;

#define ll long long

void read_adj_list(const string& filename, thrust::host_vector<ll>& h_c, thrust::host_vector<ll>& h_r) {
    ifstream file(filename);
    string line;

    ll row_pointer = 0;
    h_r.push_back(row_pointer);

    while (getline(file, line)) {
        istringstream iss(line);
        ll neighbor;

        while (iss >> neighbor) {
            h_c.push_back(neighbor);
        }

        row_pointer = h_c.size();

        h_r.push_back(row_pointer);
    }
    h_r.pop_back();
}

__global__
void bfs(ll *d_c, ll *d_r, ll *d_dist, bool *d_change, ll *d_n){
    ll thread  = blockIdx.x * blockDim.x + threadIdx.x;

    if(thread >= *d_n) return;

    for(ll i = d_r[thread]; i < d_r[thread+1]; i++){
        ll u = d_c[i];

        if(d_dist[u] > d_dist[thread]+1){
            d_dist[u] = d_dist[thread]+1;
            *d_change = 1;
        }
    }
}

string get_arg(int argc, char *argv[], string arg_name){
    for(int i = 1; i < argc; i++){
        string arg_i = argv[i];
        if(arg_i == arg_name && i+1 < argc){
            string arg = argv[i+1];
            return arg;
        }
    }
    return "";
}

int main(int argc, char *argv[]){
    int block_size = 1024;
    string graph_name = "100K";

    string arg;
    int temp;

    arg = get_arg(argc,argv,"-bs");
    if(arg != "" && (temp=stoi(arg))!=0) block_size = temp;

    arg = get_arg(argc,argv,"-g");
    if(arg != "") graph_name = arg;

    ios_base::sync_with_stdio(false);
    cin.tie(NULL);
    cout.tie(NULL);

    string filename = "connected_graph_"+graph_name+".txt";
    string output_file_name = "output_"+graph_name+"_"+to_string(block_size)+".txt";

    thrust::host_vector<ll> h_c, h_r;

    read_adj_list(filename, h_c, h_r);


    ll n = h_r.size();
    ll src = 0;

    // Some dummy vector to wake up device
    thrust::device_vector<int> dummy_vec (1000000, 1);

    output_file.close();

    thrust::device_vector<ll> d_c(h_c);

    thrust::device_vector<ll> d_r(h_r);

    thrust::device_vector<ll> d_dist(n, n-1);
    d_dist[src] = 0;

    ll *d_n;
    hipMalloc((void**)&d_n , sizeof(ll));
    hipMemcpy(d_n, &n, sizeof(ll), hipMemcpyHostToDevice);

    bool change;

    bool *d_change;
    hipMalloc((void**)&d_change , sizeof(bool));

    auto stop_time_cpy = chrono::high_resolution_clock::now();
    auto duration_cpy = chrono::duration_cast<chrono::microseconds>(stop_time_cpy - start_time_cpy);

    auto start_time_kernel = chrono::high_resolution_clock::now();

    do {
        change = 0;
        hipMemcpy(d_change, &change, sizeof(bool), hipMemcpyHostToDevice);

        bfs<<<(n/block_size)+1, block_size>>>(thrust::raw_pointer_cast(d_c.data()), thrust::raw_pointer_cast(d_r.data()), thrust::raw_pointer_cast(d_dist.data()), d_change, d_n);
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();
        if (err) {
            cerr << "Error: " << hipGetErrorString(err) << "\n";
            return 1;
        }

        hipMemcpy(&change, d_change, sizeof(bool), hipMemcpyDeviceToHost);
    } while(change);

    auto stop_time_kernel = chrono::high_resolution_clock::now();
    auto duration_kernel = chrono::duration_cast<chrono::microseconds>(stop_time_kernel - start_time_kernel);

    ofstream output_file(output_file_name);

    if (!output_file.is_open()) {
        cerr << "Error: Unable to open output file." << endl;
        return 1;
    }

    output_file << "BFS Execution Time Kernel: " << duration_kernel.count()/1000.0 << " milliseconds" << endl;

    output_file << "BFS Execution Time FULL: " << duration_kernel.count()/1000.0+duration_cpy.count()/1000.0 << " milliseconds" << endl;

    thrust::device_vector<ll> dist(d_dist.begin(), d_dist.end());

    for (ll i = 0; i < n; ++i) {
        output_file << "Distance to node " << i << ": " << dist[i] << endl;
    }

    output_file.close();

    hipFree(d_n);
    hipFree(d_change);

    return 0;
}
